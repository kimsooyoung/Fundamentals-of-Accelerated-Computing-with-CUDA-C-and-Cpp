
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

void firstParallel()
{
  printf("This should be running in parallel.\n");
}

__global__  void firstParallel_GPU()
{
  printf("This should be running in GPU parallel.\n");
}

int main()
{
  /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */

  firstParallel();
  firstParallel_GPU<<<5, 5>>>();
  hipDeviceSynchronize();

  /*
   * Some code is needed below so that the CPU will wait
   * for the GPU kernels to complete before proceeding.
   */

}
